#include "hip/hip_runtime.h"
//=============================================================================
//  Heavily based on the work of Peter Shirley and Roger Allen
//
//=============================================================================

#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <chrono>
#include <iostream>
#include "camera.cuh"
#include "hitable_list.cuh"
#include "material.cuh"
#include "sphere.cuh"

/* Constants */

#define MAX_RECURSION 20

/* Macros */

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

/* Variable declaration */

/* Function prototypes */

// Check for error in cuda funtion calls
void check_cuda(hipError_t result,
                char const* const func,
                const char* const file,
                int const line);
// Determines the color of the point where the ray hits
__device__ vec3 color(const ray& r, hitable* world, hiprandState rand_state);
// Initializes the random state for all the pixels
__global__ void rand_state_init(int width,
                                int height,
                                hiprandState* local_state);
// Creates the image to draw
__global__ void render(vec3* fb,
                       int width,
                       int height,
                       int samples,
                       camera** cam,
                       hitable** world,
                       hiprandState* rand_state);
// Creates the camera and geometry objects in the GPU memory
__global__ void create_world(hitable** list,
                             int count,
                             hitable** world,
                             camera** cam,
                             float fov,
                             const vec3 pos,
                             const vec3 look_at,
                             int width,
                             int height);
// Frees the world in the only possible way, destroying it :)
__global__ void destroy_world(hitable** list,
                              size_t count,
                              hitable** world,
                              camera** cam);

/* Main Program */

int main(int argc, char const* argv[])
{
    int width = 384;
    int height = 216;
    int pixels = width * height;
    int samples = 200;   // number of samples per pixel
    int tx = 8, ty = 8;  // threads

    size_t fb_size = pixels * sizeof(vec3);  // image size

    // allocate shared memory (CPU & GPU)
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // random state
    hiprandState* rand_state_d;
    checkCudaErrors(
        hipMalloc((void**)&rand_state_d, pixels * sizeof(hiprandState)));

    // array of objects to hit
    size_t hitable_count = 2;
    hitable** list_d;
    checkCudaErrors(
        hipMalloc((void**)&list_d, hitable_count * sizeof(hitable*)));

    // list of objects to hit
    hitable** world_d;
    checkCudaErrors(hipMalloc((void**)&world_d, sizeof(hitable*)));

    // Camera
    camera** camera_d;
    checkCudaErrors(hipMalloc((void**)&camera_d, sizeof(camera*)));

    // build the world
    create_world<<<1, 1>>>(list_d, hitable_count, world_d, camera_d, 90.0,
                           vec3(0, 2, 3), vec3(0, 0, 0), width, height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(width / tx + 1, height / ty + 1);
    dim3 threads(tx, ty);
    rand_state_init<<<blocks, threads>>>(width, height, rand_state_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, width, height, samples, camera_d, world_d,
                                rand_state_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n"
              << width << " " << height << std::endl
              << "255" << std::endl;
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel = j * width + i;
            int ir = int(255.99f * fb[pixel].r());
            int ig = int(255.99f * fb[pixel].g());
            int ib = int(255.99f * fb[pixel].b());
            std::cout << ir << " " << ig << " " << ib << std::endl;
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    destroy_world<<<1, 1>>>(list_d, hitable_count, world_d, camera_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(camera_d));
    checkCudaErrors(hipFree(world_d));
    checkCudaErrors(hipFree(list_d));
    checkCudaErrors(hipFree(rand_state_d));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();

    return 0;
}

/* Function definitions */

void check_cuda(hipError_t result,
                char const* const func,
                const char* const file,
                int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " at " << file << ":" << line << " '" << func << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_state)
{
    ray cur_ray = r;
    vec3 cur_att = vec3(1.0f, 1.0f, 1.0f);
    hit_record rec;
    for (int i = 0; i < MAX_RECURSION; i++) {
        if ((*world)->hit(cur_ray, 0.001f, MAXFLOAT, rec)) {
            ray scattered;     // output parameter
            vec3 attenuation;  // output parameter
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered,
                                     local_state)) {
                cur_att *= attenuation;
                cur_ray = scattered;
            } else {
                return vec3(0.0f, 0.0f, 0.0f);
            }
        } else {
            vec3 unit_dir = unit_vector(cur_ray.direction());
            float t = 0.5 * (unit_dir.y() + 1.0f);
            return lerp(t, vec3(1.0f, 1.0f, 1.0f), vec3(0.5f, 0.7f, 1.0f));
        }
    }
    return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void rand_state_init(int width, int height, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;
    int pixel = j * width + i;
    hiprand_init(0, pixel, 0, &rand_state[pixel]);
}

__global__ void render(vec3* fb,
                       int width,
                       int height,
                       int samples,
                       camera** cam,
                       hitable** world,
                       hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;
    int pixel = j * width + i;
    float u, v;
    hiprandState local_state = rand_state[pixel];
    vec3 col(0.0f, 0.0f, 0.0f);

    for (int k = 0; k < samples; k++) {
        u = float(i + hiprand_uniform(&local_state)) / float(width);
        v = float(j + hiprand_uniform(&local_state)) / float(height);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world, &local_state);
    }

    rand_state[pixel] = local_state;
    col /= float(samples);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel] = col;
}

__global__ void create_world(hitable** list,
                             int count,
                             hitable** world,
                             camera** cam,
                             float fov,
                             const vec3 pos,
                             const vec3 look_at,
                             int width,
                             int height)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // instantiate only once
        list[0] =
            new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f, new dielectric(2.3f));
        list[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f,
                             new diffuse(vec3(0.1f, 0.5f, 0.7f)));
        *world = new hitable_list(list, count);
        *cam = new camera(fov, float(width) / float(height), pos, look_at);
    }
}

__global__ void destroy_world(hitable** list,
                              size_t count,
                              hitable** world,
                              camera** cam)
{
    for (int i = 0; i < count; i++) {
        delete ((sphere*)list[i])->mat_ptr;
        delete list[i];
    }
    delete *world;
    delete *cam;
}
