#include "hip/hip_runtime.h"
//=============================================================================
//  Heavily based on the work of Peter Shirley and Roger Allen
//
//=============================================================================

#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <chrono>
#include <iostream>
#include "camera.cuh"
#include "hitable_list.cuh"
#include "material.cuh"
#include "sphere.cuh"
#include "moving_sphere.cuh"

/* Constants */

#define MAX_RECURSION 20
#define MEAN 0.0f
#define STD 0.3f

/* Macros */

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
#define cudaNormalDistribution(mean, std, state) \
    (hiprand_normal(state) * float(std)) + float(mean)

/* Variable declaration */

/* Function prototypes */

// Check for error in cuda funtion calls
void check_cuda(hipError_t result,
                char const* const func,
                const char* const file,
                int const line);
// Determines the color of the point where the ray hits
__device__ vec3 color(const ray& r, hitable* world, hiprandState rand_state);
// Initializes the random state for all the pixels
__global__ void rand_state_init(int width,
                                int height,
                                hiprandState* local_state);
// Creates the image to draw
__global__ void render(vec3* fb,
                       int width,
                       int height,
                       int samples,
                       camera** cam,
                       hitable** world,
                       hiprandState* rand_state);
// Creates the camera and geometry objects in the GPU memory
__global__ void create_world(hitable** list,
                             int count,
                             hitable** world,
                             camera** cam,
                             float fov,
                             const vec3 pos,
                             const vec3 look_at,
                             int width,
                             int height,
                             hiprandState* rand_state);
// Frees the world in the only possible way, destroying it :)
__global__ void destroy_world(hitable** list,
                              size_t count,
                              hitable** world,
                              camera** cam);

/* Main Program */

int main(int argc, char const* argv[])
{
    int width = 640;
    int height = 480;
    int pixels = width * height;
    int samples = 100;   // number of samples per pixel
    int tx = 8, ty = 8;  // threads

    size_t fb_size = pixels * sizeof(vec3);  // image size

    // allocate shared memory (CPU & GPU)
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // random states
    hiprandState* rand_state_d;
    hiprandState *world_rand_state_d;
    // for image rendering
    checkCudaErrors(
        hipMalloc((void**)&rand_state_d, pixels * sizeof(hiprandState)));
    // for world creation
    checkCudaErrors(
        hipMalloc((void**)&world_rand_state_d, sizeof(hiprandState)));

    // array of objects to hit
    size_t hitable_count = 4;
    hitable** list_d;
    checkCudaErrors(
        hipMalloc((void**)&list_d, hitable_count * sizeof(hitable*)));

    // list of objects to hit
    hitable** world_d;
    checkCudaErrors(hipMalloc((void**)&world_d, sizeof(hitable*)));

    // Camera
    camera** camera_d;
    checkCudaErrors(hipMalloc((void**)&camera_d, sizeof(camera*)));

    // build the world
    create_world<<<1, 1>>>(list_d, hitable_count, world_d, camera_d, 90.0f,
                           vec3(0.0f, 0.0f, 1.0f), vec3(0.0f, 0.0f, -1.0f), width, height, world_rand_state_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(width / tx + 1, height / ty + 1);
    dim3 threads(tx, ty);
    rand_state_init<<<blocks, threads>>>(width, height, rand_state_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, width, height, samples, camera_d, world_d,
                                rand_state_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n"
              << width << " " << height << std::endl
              << "255" << std::endl;
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel = j * width + i;
            int ir = int(255.99f * fb[pixel].r());
            int ig = int(255.99f * fb[pixel].g());
            int ib = int(255.99f * fb[pixel].b());
            std::cout << ir << " " << ig << " " << ib << std::endl;
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    destroy_world<<<1, 1>>>(list_d, hitable_count, world_d, camera_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(camera_d));
    checkCudaErrors(hipFree(world_d));
    checkCudaErrors(hipFree(list_d));
    checkCudaErrors(hipFree(rand_state_d));
    checkCudaErrors(hipFree(world_rand_state_d));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();

    return 0;
}

/* Function definitions */

void check_cuda(hipError_t result,
                char const* const func,
                const char* const file,
                int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " at " << file << ":" << line << " '" << func << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_state)
{
    ray cur_ray = r;
    vec3 cur_att = vec3(1.0f, 1.0f, 1.0f);
    hit_record rec;
    ray scattered;     // output parameter
    vec3 attenuation;  // output parameter
    for (int i = 0; i < MAX_RECURSION; i++) {
        if ((*world)->hit(cur_ray, 0.001f, MAXFLOAT, rec)) {
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered,
                                     local_state)) {
                cur_att *= attenuation;
                cur_ray = scattered;
            } else {
                return vec3(0.0f, 0.0f, 0.0f);
            }
        } else {
            vec3 unit_dir = unit_vector(cur_ray.direction());
            float t = 0.5 * (unit_dir.y() + 1.0f);
            return cur_att * lerp(t, vec3(1.0f, 1.0f, 1.0f), vec3(0.5f, 0.7f, 1.0f));
        }
    }
    return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void rand_state_init(int width, int height, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;
    int pixel = j * width + i;
    hiprand_init(0, pixel, 0, &rand_state[pixel]);
}

__global__ void render(vec3* fb,
                       int width,
                       int height,
                       int samples,
                       camera** cam,
                       hitable** world,
                       hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;
    int pixel = j * width + i;
    float u, v;
    hiprandState local_state = rand_state[pixel];
    vec3 col(0.0f, 0.0f, 0.0f);

    for (int k = 0; k < samples; k++) {
        /* u = float(i + cudaNormalDistribution(MEAN, STD, &local_state)) / float(width); */
        u = float(i + hiprand_uniform(&local_state)) / float(width);
        /* v = float(j + cudaNormalDistribution(MEAN, STD, &local_state)) / float(height); */
        v = float(j + hiprand_uniform(&local_state)) / float(height);
        ray r = (*cam)->get_ray(u, v, &local_state);
        col += color(r, world, &local_state);
    }

    rand_state[pixel] = local_state;
    col /= float(samples);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel] = col;
}

__global__ void create_world(hitable** list,
                             int count,
                             hitable** world,
                             camera** cam,
                             float fov,
                             const vec3 pos,
                             const vec3 look_at,
                             int width,
                             int height,
                             hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // instantiate only once
        hiprand_init(0, 0, 0, rand_state);
        hiprandState local_state = *rand_state;
        list[0] =
            new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f, new diffuse(vec3(0.1f, 0.2f, 0.5f)));
        list[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f,
                             new diffuse(vec3(0.8f, 0.8f, 0.0f)));
        list[2] =
            new moving_sphere(vec3(-1.0f, 0.0f, -1.0f), vec3(-1.0f, 0.3f, -1.0f), 0.5f, 0.0f, 1.0f, new specular(random_canonical(&local_state), 0.0f));
        list[3] =
            new sphere(vec3(1.0f, 0.0f, -1.0f), 0.5f, new dielectric(2.3f));
        *world = new hitable_list(list, count);
        *cam = new camera(fov, float(width) / float(height), pos, look_at, 0.0f, 0.5f);
    }
}

__global__ void destroy_world(hitable** list,
                              size_t count,
                              hitable** world,
                              camera** cam)
{
    for (int i = 0; i < count; i++) {
        delete ((sphere*)list[i])->mat_ptr;
        delete list[i];
    }
    delete *world;
    delete *cam;
}
